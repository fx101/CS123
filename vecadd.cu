#include "hip/hip_runtime.h"
/*
 * Vector Addition Kernel
 * From NVIDIA CUDA by Example
 */

__global__ void add(int *a, int *b, int *c)
{
	int tID = blockIdx.x;
	if (tID < N)
	{
		c[tID] = a[tID] + b[tID];
	}
}




