/*
 *Alfredo Luque
 *CUDA 3-layer MLP
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <nn.h>
#include <array_reduction.h>
#include <csv_v2.h>

using namespace std;

float* initW(unsigned int nW)
{
	float * weights;
	weights = new float[nW];
	srand(time(0));
	for(int i = 0; i < nW ; i++)
	{
		weights[i] =  ((float)((rand() % 1001)-1000))/1000.0;
	}
	return weights;
}

void sumArray(float* errIn, float* partsum , float* error)
{
	//sum errors to blockdim numbers
	sumreduce<<<N/blockSize,blockSize>>>(errIn,partsum,N); //reduces into redBlocks floats
	sumreduce<<<1,blockSize>>>(partsum,error,N); //reduces to final sum
}

void sumVec(float* a , float* b, float* c) //a+b=c
{
	// Fill Arrays
	for (int i = 0; i < N; i++)
		hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
		add<<<N,1>>>(dev_a, dev_b, dev_c);
		hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);
}

float* loadInputs(float* dev_array, bool inputfile)
{
  if(inputfile)
  {
	  io::CSVReader<3> in("inputs.csv"); //just rename the file inputs.csv
  }
  else
  {
	  io::CSVReader<3> in("outputs.csv"); //just rename the file outputs.csv
  }
  in.read_header(io::ignore_extra_column, "Weather", "OnTime", "Delay/Out");
  std::string vendor; int size; double speed;
  while(in.read_row(vendor, size, speed)){
	  for(int i = 0 ; i < IN)
	  {
		  hipMemcpy(row[i], dev_in , N*sizeof(float), hipMemcpyHostToDevice);
	  }
  }
}

int main(void)
{
	//load training data

	//Finalize block dimensions
	//Initialize Weights
	unsigned int nW = (unsigned int)((IN+1)*HN*(LAYERS-1)); //One bias per hidden and output layer
	float* wSeeds;
	wSeeds = initW(nW);
	float* dev_w;
	hipMalloc((void**)&dev_w,(sizeof(float)*nW));
	hipMemcpy(wSeeds , dev_w , sizeof(float)*nW , hipMemcpyHostToDevice);

	//Generate Device Error Array
	float* dev_errIn;
	float* dev_errPartSum;
	float* dev_error;
	hipMalloc((void**)&dev_errIn,(sizeof(float)*N));
	hipMalloc((void**)&dev_errPartSum,(sizeof(float)*N));
	hipMalloc((void**)&dev_error,sizeof(float));
	//Generate Device Gross Update Array
	float* dev_grossUp;
	hipMalloc((void**)&dev_grossUp,N*sizeof(float)*nW);
	//Generate Prev Update Arrays (Momentum)
	float* dev_prevUp;
	hipMalloc((void**)&dev_prevUp,sizeof(float)*nW);

	float* dev_tdi
	hipMalloc((void**)&dev_prevUp,sizeof(float)*IN*N);
	float* dev_tdo;
	hipMalloc((void**)&dev_prevUp,sizeof(float)*ON*N); //usually just size N*sizeof(float) since I have one output neuron
	loadinputs(dev_tdi,true);
	loadinputs(dev_tdo,false);
	//Iterate Backpropagation!
	for(int i = 0 ; i < EPOCHS ; i++)
	{
		kernBackProp<<<N,HN>>>(dev_tdi, dev_tdo, dev_w ,dev_grossUp , dev_prevUp dev_errIn);
		sumArray(dev_errIn, dev_error); //we can output this somewhere to create convergence charts
		for(int j = 0 ; j < N ; j++)
		{
			sumVec(&grossUp[i] , &grossUp[i+1] , &dev_weights); //compute net changes
		}
		hipMemcpy(dev_weights, wei)
	}
	hipMemcpy(dev_weights, weights, (sizeof(float)*nW) , hipMemcpyDeviceToHost ); //retrieve weights (that's what we're after!).
	return 0;
}
