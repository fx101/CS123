#include "hip/hip_runtime.h"
/*
 *Alfredo Luque
 *CUDA 3-layer MLP
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <nn.h>

using namespace std;

float* initW(unsigned int nW)
{
	float * weights;
	weights = new float[nW];
	srand(time(0));
	for(int i = 0; i < nW ; i++)
	{
		weights[i] =  ((float)((rand() % 1001)-1000))/1000.0;
	}
	return weights;
}



int main(void)
{
	//Initialize Weights
	unsigned int nW = (unsigned int)(pow(IN,2.0)*2.0); //max dim for square weight matrix
	float* wSeeds;
	wSeeds = initW(nW);
	float* dev_wSeeds;
	hipMalloc((void**)&dev_wSeeds,(sizeof(float)*nW));
	hipMemcpy(wSeeds , dev_wSeeds , sizeof(float)*nW , hipMemcpyDeviceToHost);

	//load data
	return 0;
}
