/*
 *Alfredo Luque
 *CUDA 3-layer MLP
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <nn.h>
#include <array_reduction.h>

using namespace std;

float* initW(unsigned int nW)
{
	float * weights;
	weights = new float[nW];
	srand(time(0));
	for(int i = 0; i < nW ; i++)
	{
		weights[i] =  ((float)((rand() % 1001)-1000))/1000.0;
	}
	return weights;
}

void sumArray(unsigned int numBlocks)
{
	//sum errors to blockdim numbers
	sumreduce<<redBlocks,blockSize>>(dev_errIn,dev_PartSum); //reduces into redBlocks floats
	sumreduce<<1,redBlocks>>(dev_PartSum,dev_error); //reduces to final sum
}


int main(void)
{
	//load training data

	//Finalize block dimensions
	unsigned int redBlocks = N/blockSize;
	//Initialize Weights
	unsigned int nW = (unsigned int)(IN*HN*(LAYERS-1));
	float* wSeeds;
	wSeeds = initW(nW);
	float* dev_w;
	hipMalloc((void**)&dev_w,(sizeof(float)*nW));
	hipMemcpy(wSeeds , dev_w , sizeof(float)*nW , hipMemcpyDeviceToHost);

	//Generate Device Error Array
	float* dev_errIn;
	float* dev_errPartSum;
	float* dev_error;
	hipMalloc((void**)&dev_errIn,(sizeof(float)*N));
	hipMalloc((void**)&dev_errPartSum,(sizeof(float)*N));
	hipMalloc((void**)&dev_error,sizeof(float));
	//propagate network
	actNodeCol<<N,HN>>(dev_tdi, dev_tdo, dev_w , dev_errIn);

	return 0;
}
