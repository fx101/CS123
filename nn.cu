#include "hip/hip_runtime.h"
/*
 * Kernels For Training ANN
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <nn.h>

__device__ float sigmoid(float *x)
{
	return (1.0/(1.0 + __expf(-*x)));
}

__device__ float DSigmoid(float *x)
{
	return sigmoid(x)*(1-sigmoid(x));
}

__device__ float logit(float *x)
{
	return __logf(*x) - __logf(1.0-*x);
}

__device__ void initActMat(float * ins , float * actMatrix)
{
	for(int i=0 ; i < IN ; i++)
	{
		actMatrix[i] = ins[threadIdx.x*IN + i];
	}
}

__device__ void sliceData(float* glob_data , float* dev_data)
{
	for(int i=0 ; i < IN ; i++)
		{
			dev_data[i] = glob_data[blockIdx.x*IN + i];
		}
}

__global__ void kernBackProp(float* ins, float* outs, float* weights, float* updates)
{
	/*
	 * Weights are flattened in the following form:
	 * Weight array is subdivided into (Layers-1) partitions of size IN^2 that correspond to layers
	 * Each of these partitions is subdivided into HN partitions of size IN
	 *
	 */
	__shared__ float inputs[IN]; //on-chip subsection of inputs
	__shared__ float activations[IN*LAYERS];
	__shared__ float dev_weights[(IN+1)*HN*(LAYERS-1)];
	__shared__ float partSums[IN*LAYERS];

	*dev_weights = *weights; //on-chip copy of weights

	sliceData(ins, inputs);
	initActMat(inputs,activations);

	//Initialize Inputs
	for(int i = 0; i < IN ; i++)
	{
		activations[IN+i] = sigmoid(&activations[i]);
	}

	//Weighted Sum to Hidden Neuron
	__syncthreads();
	float hnSum = 0.0;
	for(int i = 0; i < IN ; i++)
	{
		hnSum += dev_weights[(IN+1)*threadIdx.x + i] * activations[IN+i];
	}
	hnSum += dev_weights[(IN+1) * threadIdx.x + IN]; //hidden bias

	//Store Output from Hidden Neuron
	activations[2*IN + threadIdx.x] = sigmoid(&hnSum);

	//Weighted Sum to Output Neuron
	__syncthreads();
	if(threadIdx.x < ON)
	{
		float onSum = 0.0;
		for(int i = 0; i < HN ; i++)
		{
			onSum += dev_weights[HN*(IN+1) + (IN+1)*threadIdx.x + i] * activations[2*IN+i];
		}
		onSum += dev_weights[HN*(IN+1) + (IN+1)*threadIdx + IN]; //output bias
		//Output Neuron Activations
		activations[3*IN + threadIdx.x] = sigmoid(&onSum);

		//Sq Error
		for(int i = 0; i < ON ; i++)
		{
			errors[blockIdx.x] += (outs[i]-activations[3*IN + i])*(outs[i]-activations[3*IN + i]);
		}
		errors[blockIdx.x] *= 0.5;
	}

	//Backpropagate



}

