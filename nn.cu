#include "hip/hip_runtime.h"
/*
 * Kernels For Training ANN
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <nn.h>

__device__ float sigmoid(float *x)
{
	return (1.0/(1.0 + __expf(-*x)));
}

__device__ void initActMat(float * ins , float * actMatrix)
{
	for(int i=0 ; i < IN ; i++)
	{
		actMatrix[i] = ins[threadIdx.x*IN + i];
	}
}

__device__ void sliceData(float* glob_data , float* dev_data)
{
	for(int i=0 ; i < IN ; i++)
		{
			dev_data[i] = glob_data[blockIdx.x*IN + i];
		}
}

__global__ void actNodeCol(float* ins, float* outs, float* weights, float* errors)
{
	//Setup per-block flattened matrices
	__shared__ float inputs[IN]; //on-chip subsection of inputs
	__shared__ float activations[IN*LAYERS];
	__shared__ float dev_weights[IN*IN*(LAYERS-1)];
	*dev_weights = *weights; //on-chip copy of weights

	sliceData(ins, inputs);
	initActMat(inputs,activations);

	//Initialize Inputs
	for(int i = 0; i < IN ; i++)
	{
		activations[IN+i] = sigmoid(&activations[i]);
	}

	//Weighted Sum to Hidden Neuron
	__syncthreads();
	float hnSum = 0.0;
	for(int i = 0; i < IN ; i++)
	{
		hnSum += dev_weights[IN*threadIdx.x + i] * activations[IN+i];
	}
	//Store Output from Hidden Neuron
	activations[2*IN + threadIdx.x] = sigmoid(&hnSum);

	//Weighted Sum to Output Neuron
	__syncthreads();
	if(threadIdx.x < ON)
	{
		float onSum = 0.0;
		for(int i = 0; i < HN ; i++)
		{
			onSum += dev_weights[IN*threadIdx.x + i] * activations[2*IN+i];
		}
		//Store Output from Output Neuron
		activations[3*IN + threadIdx.x] = sigmoid(&onSum);
		//Calculate Squared Error (Assumes errors initialized to 0)
		for(int i = 0; i < ON ; i++)
		{
			errors[blockIdx.x] += __powf(outs[i]-activations[3*IN + i] , 2.0);
		}
	}
}
