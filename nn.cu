#include "hip/hip_runtime.h"
/*
 * Kernels For Training ANN
 * Optimized for One Hidden Layer
 * IMPORTANT: Arbitrary Layer Connectivity Requires Nesting in additional loop. Use NVCC loop unrolling flags to guarantee good performance.
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <nn.h>



__device__ void initActMat(float * ins , float * actMatrix)
{
	for(int i=0 ; i < IN ; i++)
	{
		actMatrix[i] = ins[threadIdx.x*IN + i];
	}
}

__device__ void sliceData(float* glob_data , float* dev_data, bool input)
{
	if(input == true) //Slice Inputs
	{
	for(int i=0 ; i < IN ; i++)
		{
			dev_data[i] = glob_data[blockIdx.x*IN + i];
		}

	}
	else //Slice Outputs
	{
		for(int i=0 ; i < ON ; i++)
		{
			dev_data[i] = glob_data[blockIdx.x*ON + i];
		}

	}
}

__global__ void kernBackProp(float* ins, float* outs, float* weights, float* grossUpdates, float* prevNetUp, float* outerrors)
{
	/*
	 * Weights are flattened in the following form:
	 * Weight array is subdivided into (Layers-1) partitions of size HN*(IN+1) that correspond to layers
	 * Each of these partitions is subdivided into HN partitions of size IN+1 (up to IN inputs and 1 bias)
	 *
	 */
	__shared__ float inputs[IN]; //on-chip subsection of inputs
	__shared__ float outputs[ON];
	__shared__ float activations[IN*LAYERS];
	__shared__ float dev_weights[(IN+1)*HN*(LAYERS-1)];
	__shared__ float partSums[IN*LAYERS];
	__shared__ float deltas[(IN+1)*HN*(LAYERS-1)];
	__shared__ float outdeltas[ON];

	*dev_weights = *weights; //on-chip copy of weights

	sliceData(ins, inputs,true);
	sliceData(outs, outputs,false);
	initActMat(inputs,activations);

	//Initialize Inputs
	for(int i = 0; i < IN ; i++)
	{
		activations[IN+i] = sigmoid(&activations[i]);
	}

	//Weighted Sum to Hidden Neuron
	__syncthreads();
	float hnSum = 0.0;
	for(int i = 0; i < IN ; i++)
	{
		hnSum += dev_weights[(IN+1)*threadIdx.x + i] * activations[IN+i];
	}
	hnSum += dev_weights[(IN+1) * threadIdx.x + IN]; //hidden bias

	//Store Output from Hidden Neuron
	activations[2*IN + threadIdx.x] = sigmoid(&hnSum);

	//Weighted Sum to Output Neuron
	__syncthreads();
	if(threadIdx.x < ON)
	{
		float onSum = 0.0;
		for(int i = 0; i < HN ; i++)
		{
			onSum += dev_weights[HN*(IN+1) + (IN+1)*threadIdx.x + i] * activations[2*IN+i];
		}
		onSum += dev_weights[HN*(IN+1) + (IN+1)*threadIdx + IN]; //output bias

		//Output Neuron Activations
		activations[3*IN + threadIdx.x] = sigmoid(&onSum);
		//Output Deltas
		outdeltas[threadIdx.x] = outputs[threadIdx.x]-activations[3*IN + threadIdx.x];

		//Weight Changes Hidden:Output
		for(int i = 0 ; i < (HN+1) ; i++)
		{
			deltas[HN*(IN+1) + (IN+1)*threadIdx.x + i] = dev_weights[HN*(IN+1) + (IN+1)*threadIdx.x + i]*outdeltas[threadIdx.x]*DSigmoid(&activations[3*IN + threadIdx.x]);

			//No getting around the insane indices. Apologies to the brave soul who reads this.
			//Basically... change = -learningrate*delta*activation + momentum*previousweightchange
			grossUpdates[BlockIdx.x*((IN+1)*HN*(LAYERS-1))+(HN*(IN+1) + (IN+1)*threadIdx.x + i)]=(-1.0)*(LR*activations[HN*(IN+1) + (IN+1)*threadIdx.x + i]*deltas[HN*(IN+1) + (IN+1)*threadIdx.x + i] +(prevNetUp[HN*(IN+1) + (IN+1)*threadIdx.x + i]*MOM));
		}
	}
	__syncthreads();
	//Weight Changes From Hidden to Input Layer
	for(int i = 0 ; i < IN+1 ; i++)
	{
		for(int i = 0; i < HN ; i++) //sum over all hidden neurons
		{
			deltas[(IN+1)*threadIdx.x + i] = dev_weights[(IN+1)*threadIdx.x + i]*deltas[HN*(IN+1) + (IN+1)*threadIdx.x + i]*DSigmoid(&activations[2*IN + threadIdx.x]);
			grossUpdates[BlockIdx.x*((IN+1)*HN*(LAYERS-1))+((IN+1)*threadIdx.x + i)] = (-1.0)*(LR*activations[2*IN + threadIdx.x]*deltas[(IN+1)*threadIdx.x + i])+(MOM*prevNetUp[(IN+1)*threadIdx.x + i]);
		}
	}

	//We store output sum squared at end of kernel to mitigate warp divergence
	if(threadIdx.x == 0)
	{
		for(int i = 0 ; i < ON ; i++)
		{
			outErrors[BlockIdx.x] = (outputs[i]-activations[3*IN + i])*(outputs[i]-activations[3*IN + i]);
		}
	}
}

